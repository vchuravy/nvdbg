
#include <hip/hip_runtime.h>
__global__ void add(float* out, float* X, float* Y) {
  long idx = threadIdx.x;
  out[idx] = X[idx] +Y[idx];
}
